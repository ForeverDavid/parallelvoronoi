//adapted from: http://docs.nvidia.com/cuda/hiprand/host-api-overview.html#host-api-example

#include "cudarand.cuh"

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
namespace cudaRand {
/**
 * Fill an array with random floats using the CURAND function.
 * \param devData The chunk of memory you want to fill with floats within the range (0,1]
 * \param n The size of the chunk of data
 */
int randFloats(float *&devData, const size_t n)
{
    // The generator, used for random numbers
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

    // Set seed to be the current time (note that calls close together will have same seed!)
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

    // Generate n floats on device
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
    // Cleanup
    CURAND_CALL(hiprandDestroyGenerator(gen));
    return EXIT_SUCCESS;
}
}
