#include "hip/hip_runtime.h"
#include "CudaSolver.cuh"
#include "CudaSolver.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>

//----------------------------------------------------------------------------------------------------------------------
__device__ uint d_distSquared(uint _p1x, uint _p1y, uint _p2x, uint _p2y)
{
    int xd = _p2x-_p1x;
    int yd = _p2y-_p1y;
    return (xd * xd) + (yd * yd);
}
//----------------------------------------------------------------------------------------------------------------------
__device__ uint * d_checkDistance(uint _x, uint _y, uint _w, uint _h, uint _res,
                                        real *_Xpositions, real *_Ypositions,
                                        uint *_excScan, uint *_cellOcc)
{
    uint dist = INT32_MAX;
    uint colIDX = -1;

    real x021 = _x/_w;
    real y021 = _y/_h;

    uint xfrac = floor(x021 * _res);
    uint yfrac = floor(y021 * _res);

    uint gridPos[2] = {xfrac, yfrac};

    uint scanIDX = gridPos[0] * _res + gridPos[1];

    uint startIndex = _excScan[scanIDX];
    uint endIndex = startIndex + _cellOcc[scanIDX];

    uint d;

    //Get reduced set of cells
    for(uint i = startIndex; i < endIndex; i++)
    {
        d = d_distSquared(_x, _y, _Xpositions[i]*_w, _Ypositions[i]*_h);

        if(d < dist)
        {
            dist = d;
            colIDX = i;
        }
    }
    uint ret[2] = {d, colIDX};
    return ret;
}
//----------------------------------------------------------------------------------------------------------------------
__global__ void g_calculateVoronoiDiagram_NN(uint _cellCount, uint _w, uint _h, uint _res,
                                             real *_Xpositions, real *_Ypositions,
                                             uint *_hash, uint *_excScan, uint *_cellOcc,
                                             uint* _pixelVals)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint pixCount = _w*_h;
    if(idx > pixCount)
        return;

    uint x = idx%_w;
    uint y = (idx-x)/_w;

    uint dist = INT32_MAX;
    uint colIDX = -1;

    for(int i = -1; i<2; i++)
    {
        for(int j = -1; j<2; j++)
        {
            uint neighborX = x + (i*(_w*_res));
            uint neighborY = y + (j*(_h*_res));
            uint * neighborPair = d_checkDistance(neighborX, neighborY,
                                                  _w, _h, _res,
                                                  _Xpositions, _Ypositions,
                                                  _excScan, _cellOcc);
            if(neighborPair[0] < dist)
            {
                dist = neighborPair[0];
                colIDX = neighborPair[1];
            }
        }
    }


    _pixelVals[idx] = colIDX;
}

//----------------------------------------------------------------------------------------------------------------------
__global__ void g_pointHash(uint *_hash,  const real *_Xpositions, const real *_Ypositions, const uint _res, uint _cellCount)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx > _cellCount)
        return;

    //Need to map the _Xposes from 0-1 first
    uint xfrac = floor(_Xpositions[idx] * _res);
    uint yfrac = floor(_Ypositions[idx] * _res);

    uint gridPos[2] = {xfrac, yfrac};

    _hash[idx] = gridPos[0] * _res + gridPos[1];
}
//----------------------------------------------------------------------------------------------------------------------
__global__ void g_countCellOcc(uint *_hash, uint *_cellOcc, uint _pixCount, uint _hashCellCount)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < _pixCount && _hash[idx] < _hashCellCount)
        atomicAdd(&(_cellOcc[_hash[idx]]), 1);
}
//----------------------------------------------------------------------------------------------------------------------
__global__ void g_calculateVoronoiDiagram_brute(uint _cellCount, uint _w,
                                                uint *_positions, uint* _pixelVals)
{
    //---------------------------------------------
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint dist = INT32_MAX;
    uint colIDX = -1;

    //Iterate through each cell
    //---------------------------------------------
    for(uint i = 0; i < _cellCount; i++)
    {
        //Determine the position of this pixel and calculate its distance squared from the current cell
        //---------------------------------------------
        uint x = idx%_w;
        uint y = (idx-x)/_w;
        uint d = d_distSquared(x, y, _positions[i], _positions[i+_cellCount]);
        //---------------------------------------------

        //If this is the shortest distance we have found so far save this index
        //---------------------------------------------
        if(d < dist)
        {
            dist = d;
            colIDX = i;
        }
        //---------------------------------------------
    }
    //Set this pixels colour index equal to that of the closest cell
    //---------------------------------------------
    _pixelVals[idx] = colIDX;
}
