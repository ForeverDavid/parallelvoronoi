#include "hip/hip_runtime.h"
#include "CudaSolver.cuh"
#include "CudaSolver.h"
#include "cudarand.cuh"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>

//----------------------------------------------------------------------------------------------------------------------

CUDASolver::CUDASolver()
{
}

void CUDASolver::checkCUDAErr()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
}

void CUDASolver::printCudaInfo()
{
    int runtimeVer, driverVer;
    hipRuntimeGetVersion(&runtimeVer);
    hipDriverGetVersion(&driverVer);
    std::cout<<"CUDA INFO -------------------\nRuntime Version: "<<runtimeVer<<"\nDriver Version: "<<driverVer<<"\n";
    std::cout<<"GPU INFO---------------------\n";
    //Code from https://devblogs.nvidia.com/how-query-device-properties-and-handle-errors-cuda-cc/
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device name: %s\n", prop.name);
    printf("Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    //End Citation
    std::cout<<"-----------------------------\n";
}

std::vector<vec3> CUDASolver::makeDiagram(uint _imageDimsX, uint _imageDimsY, uint _cellCount)
{
    thrust::host_vector<uint> h_cellPositions(_cellCount*2);
    h_cellPositions.reserve(_cellCount*2);

    thrust::host_vector<uint> h_cellColours(_cellCount*3);
    h_cellPositions.reserve(_cellCount*3);

    for(uint i = 0; i < _cellCount*2; i++)
    {
        if(i < _cellCount)
            h_cellPositions[i] = randNum(_imageDimsX);
        else
            h_cellPositions[i] = randNum(_imageDimsY);
    }

    for(uint i = 0; i < _cellCount*3; i++)
    {
        h_cellColours[i] = randNum(255);
    }

    thrust::device_vector<uint> d_cellColours(h_cellColours);
    uint * d_cellColours_ptr = thrust::raw_pointer_cast(&d_cellColours[0]);

    thrust::device_vector<uint> d_cellPositions(h_cellPositions);
    uint * d_cellPositions_ptr = thrust::raw_pointer_cast(&d_cellPositions[0]);

    thrust::device_vector<uint> d_results(_imageDimsX * _imageDimsY);
    uint * d_results_ptr = thrust::raw_pointer_cast(&d_results[0]);

    uint blockCount = std::ceil(_imageDimsX*_imageDimsY)/1024;

    struct timeval tim;
    double t1, t2;
    gettimeofday(&tim, NULL);
    t1=tim.tv_sec+(tim.tv_usec * 0.0000001);

    g_calculateVoronoiDiagram<<<blockCount, (_imageDimsX*_imageDimsY)/blockCount>>>(_cellCount, _imageDimsX, _imageDimsY, d_cellPositions_ptr, d_cellColours_ptr, d_results_ptr);
    checkCUDAErr();
    hipDeviceSynchronize();

    gettimeofday(&tim, NULL);
    t2=tim.tv_sec+(tim.tv_usec * 0.0000001);
    std::cout << "GPU Algorithm took: " << t2-t1 << "s for "<<_cellCount<<" cells\n";


    thrust::host_vector<uint> h_results(d_results);

    std::vector<vec3> retVec(_imageDimsX * _imageDimsY);

    for(uint i = 0; i < _imageDimsX * _imageDimsY; i++)
    {
        uint r = h_cellColours[h_results[i]];
        uint g = h_cellColours[h_results[i]+_cellCount];
        uint b = h_cellColours[h_results[i]+_cellCount+_cellCount];

        retVec[i] = vec3(r, g, b);
    }

    return retVec;
}

template<typename T>
T CUDASolver::randNum(T _max)
{
    std::random_device r;

    std::mt19937 e(r());

    std::uniform_real_distribution<> uniform_dist(0.0, _max);

    return uniform_dist(e);
}

void CUDASolver::hello()
{
    k_hello<<<4, 32>>>();
    checkCUDAErr();
    hipDeviceSynchronize();
}
